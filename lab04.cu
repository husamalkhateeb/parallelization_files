
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define N 1024
#define M 1024

__global__ void parallel_init(int **A)
{
  A[blockIdx.x][threadIdx.x] = (int)(0.5 * blockIdx.x + 0.5 * threadIdx.x + 1);
 
}

__global__ void parallel_sum(int **A, int *sum)
{
  int offset = blockDim.x;
  for (int d = offset; d >= 1; d = d / 2)
  {
    if (threadIdx.x < d)
      A[blockIdx.x][threadIdx.x] += A[blockIdx.x][threadIdx.x + d];
    __syncthreads();
  }

  if (threadIdx.x == 0)
    atomicAdd(sum,A[blockIdx.x][0]);
}

int main()
{
  int **A;
  int i;
  int * sum = 0;

 hipMallocManaged(&A,N * sizeof(int *));
  hipMallocManaged(&sum, 1*sizeof(int));
  for (i = 0; i < N; i++)
 hipMallocManaged(&A[i],M * sizeof(int));

  parallel_init<<<N, M>>>(A);
  hipDeviceSynchronize();

  parallel_sum<<<N, M / 2>>>(A, sum);
  hipDeviceSynchronize();
  printf("sum = %d\n", *sum);
  for(i=0; i<N; i++){
    hipFree(A[i]);  }
  
  hipFree(A);
  hipFree(sum);
  return 0;
}